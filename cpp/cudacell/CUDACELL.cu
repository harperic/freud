#include "hip/hip_runtime.h"
#include <ostream>
#include "CUDACELL.cuh"

using namespace std;

namespace freud { namespace cudacell {

// Part 3 of 5: implement the kernel
__global__ void computeCellList(unsigned int *p_array,
                                unsigned int *c_array,
                                unsigned int np,
                                unsigned int nc,
                                trajectory::CudaBox box,
                                Index3D& cell_idx,
                                float3 *points)
    {
    // determine particle being calculated
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < np)
        {
        // get the point
        float3 point = points[idx];
        // determine cell for idx
        float3 alpha = box.makeFraction(point);
        uint3 c;
        c.x = floorf(alpha.x * float(cell_idx.getW()));
        c.x %= cell_idx.getW();
        c.y = floorf(alpha.y * float(cell_idx.getH()));
        c.y %= cell_idx.getH();
        c.z = floorf(alpha.z * float(cell_idx.getD()));
        c.z %= cell_idx.getD();
        uint3 c_idx = cell_idx(c.x, c.y, c.z);
        p_array[idx] = idx;
        c_array[idx] = c_idx;
        }
    }

void CallCompute(unsigned int *p_array,
                 unsigned int *c_array,
                 unsigned int np,
                 unsigned int nc,
                 trajectory::CudaBox &box,
                 Index3D& cell_idx,
                 float3 *points)
    {

    // define grid and block size
    int numThreadsPerBlock = 32;
    int numBlocks = (arrSize / numThreadsPerBlock) + 1;

    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    computeCellList<<< dimGrid, dimBlock >>>( p_array, c_array, np, nc, box, cell_idx, points );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    }

void createIDXArray(unsigned int **IDXArray, size_t memSize)
    {
    hipMallocManaged(IDXArray, memSize);
    hipDeviceSynchronize();
    }

void freeIDXArray(unsigned int **IDXArray)
    {
    hipFree(IDXArray);
    }

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

}; }; // end namespace freud::cudacell
